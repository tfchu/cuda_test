#include "hip/hip_runtime.h"
/* 
onehiddenlayerperceptron.cu
onehiddenlayerperceptron

Created by Sergei Bugrov on 8/21/17.
Copyright © 2017 Sergei Bugrov. All rights reserved.

output
$ ./one_layer_perceptron
Prediction[0] : 0.060997 True Value[0] : 0.000000 Error[0] : 0.060997
Prediction[1] : 0.076193 True Value[1] : 0.000000 Error[1] : 0.076193
Prediction[2] : 0.927551 True Value[2] : 1.000000 Error[2] : -0.072449
Prediction[3] : 0.918263 True Value[3] : 1.000000 Error[3] : -0.081737
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// kernel of elementwise multiplication of 2 arrays
__global__ 
void kMatrixByMatrixElementwise(const int nThreads, const float *m1, const float *m2, float *output) {
    /*  Computes the product of two arrays (elementwise multiplication).
     Inputs:
     m1: array
     m2: array
     output: array,the results of the multiplication are to be stored here
    */
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	  {
		output[i] = m1[i] * m2[i];
	  }
}

// elementwise multiplication of 2 arrays
__device__ 
float* dMatrixByMatrixElementwise(const float *m1, const float *m2, float *output, const int width, const int height){

	kMatrixByMatrixElementwise <<< width, height >>> ( width * height, m1, m2, output );
    hipDeviceSynchronize();
    return output;
}

// kernel elementwise difference of 2 arrays
__global__ 
void kMatrixSubstractMatrix(const int nThreads, const float *m1, const float *m2, float *output) {
    /*  Computes the (elementwise) difference between two arrays
     Inputs:
     m1: array
     m2: array
     output: array,the results of the computation are to be stored here
     */

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	  {
		output[i] = m1[i] - m2[i];
	  }
}

// elementwise difference of 2 arrays
__device__ 
float* dMatrixSubstractMatrix(const float *m1, const float *m2, float *output, const int width, const int height){

	kMatrixSubstractMatrix <<< width, height >>> ( width * height, m1, m2, output );
    hipDeviceSynchronize();
    return output;
}

// kernel of device sigmoid function
__global__ 
void kSigmoid(const int nThreads, float const *input, float *output){
    /*  Computes the value of the sigmoid function f(x) = 1/(1 + e^-x).
     Inputs:
     input: array
     output: array, the results of the computation are to be stored here
    */

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	  {
		output[i] = 1.0 / (1.0 + std::exp(-input[i]));
	  }
}

// cuda version (device-side) of sigmoid function
__device__ 
void dSigmoid(float const *input, float *output, const int height, const int width){

	kSigmoid <<< height, width >>> (height * width, input, output);
	hipDeviceSynchronize();
}

// kernel of derivative of sigmoid function
__global__ 
void kSigmoid_d(const int nThreads, float const *input, float *output) {
	/*  Computes the value of the sigmoid function derivative f'(x) = f(x)(1 - f(x)),
	    where f(x) is sigmoid function.
	    Inputs:
	    input: array
	    output: array, the results of the computation are to be stored here:
	    		x(1 - x) for every element of the input matrix m1.
	*/

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	  {
		output[i] = input[i] * (1 - input[i]);
	  }
}

// derivative of sigmoid function (d: device, d: derivative)
__device__ 
float* dSigmoid_d(float const *input, float *output, const int rows, const int columns){
	kSigmoid_d <<< rows, columns >>> (rows*columns, input, output);
	hipDeviceSynchronize();
	return output;
}

// kernel of product of two matrices: m1 x m2
__global__ 
void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){
	/*  Computes the product of two matrices: m1 x m2.
	   	Inputs:
	    m1: array, left matrix of size m1_rows x m1_columns
	    m2: array, right matrix of size m1_columns x m2_columns (the number of rows in the right matrix
	    must be equal to the number of the columns in the left one)
	    output: array, the results of the computation are to be stored here:
	    		m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_columns
	    m1_rows: int, number of rows in the left matrix m1
	    m1_columns: int, number of columns in the left matrix m1
	    m2_columns: int, number of columns in the right matrix m2
	*/

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	{
	    int r = (int)i / m2_columns;		// 
	    int c = i % m2_columns;
	    float t_output = 0.f;

	    for( int k = 0; k < m1_columns; ++k ) {
	        t_output += m1[ r * m1_columns + k ] * m2[ k * m2_columns + c ];
	    }

	    output[i] = t_output;
	}
}

// product of two matrices: m1 x m2
// output is m1_rows x m2_columns
__device__ 
float* dDot(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_columns ){

	kDot <<< m1_rows, m2_columns >>> (m1_rows * m2_columns, m1, m2, output, m1_rows , m1_columns, m2_columns );
	hipDeviceSynchronize();
	return output;
}

__global__ 
void kDot_m1_m2T(const int nThreads, const float *m1, const float *m2, float *output, const int m1_columns, const int m2_rows ){
	/*  Updates the output matrix with the product of two matrices: m1 and m2 transposed.
	   	Inputs:
	    m1: array, left matrix of size m1_rows x m1_columns
	    m2: array, right matrix of size m2_rows x m1_columns (m2 transposed will be of size m1_columns x m2_rows)
	    output: array, the results of the computation are to be stored here:
	    		m1 * m2, product of two arrays m1 and m2, a matrix of size m1_rows x m2_rows
	    m1_columns: int, number of columns in the left matrix m1
	    m2_rows: int, number of rows in the left matrix m2
	*/

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	{
		int r = (int)i / m2_rows;
		int c = i % m2_rows;
		float t_output = 0.0;
		int id_T;

		for( int k = 0; k < m1_columns; ++k ) {
			id_T = c * m1_columns + k;
			t_output += m1[ r * m1_columns + k ] * m2[ id_T ];
		}

		output[i] = t_output;
	}
}

__device__ 
float* dDot_m1_m2T(const float *m1, const float *m2, float *output, const int m1_rows , const int m1_columns, const int m2_rows )
{
	kDot_m1_m2T <<< m1_rows, m2_rows >>> ( m1_rows * m2_rows, m1, m2, output, m1_columns, m2_rows );
	hipDeviceSynchronize();
	return output;
}

__global__ 
void kDot_m1T_m2(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows,
							const int m1_columns, const int m2_columns ){
	/*  Increments the output matrix with the product of two matrices: m1 transposed and m2.
	   	Inputs:
	    m1: array, left matrix of size m1_rows x m1_columns (m1 transposed will be of size m1_columns x m1_rows)
	    m2: array, right matrix of size m1_rows x m2_columns
	    output: array, the results of the computation are to be stored here:
	    		m1 * m2, product of two arrays m1 and m2, a matrix of size m1_columns x m2_columns
	    m1_rows: int, number of rows in the left matrix m1
	    m1_columns: int, number of columns in the left matrix m1
	    m2_rows: int, number of rows in the left matrix m2
	*/

	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		 i < nThreads;
		 i += blockDim.x * gridDim.x)
	{
	    int r = (int)i / m2_columns;
	    int c = i % m2_columns;
	    int id_T;
	    float t_output = 0.0;

	    for( int k = 0; k < m1_rows; ++k ) {
	    	id_T = k * m1_columns + r;
	        t_output += m1[ id_T ] * m2[ k * m2_columns + c ];
	    }

	    output[i] += t_output;
	}
}

__device__ 
void dDot_m1T_m2(const float *m1, const float *m2, float *output, const int m1_height , const int m1_width, const int m2_width )
{
	kDot_m1T_m2 <<< m1_width, m2_width >>> (m1_width * m2_width, m1, m2, output, m1_height, m1_width, m2_width );
	hipDeviceSynchronize();
}

__device__ 
void kPrintMatrix (const float* M, int h, int w) {
    /*  Prints out the input array as h x w matrix.
     Inputs:
     m: vector, matrix of size n_rows x n_columns
     h: int, number of rows in the matrix M
     w: int, number of columns in the matrix M
     */
	for (int i = 0; i < h; i++){
		for (int j = 0; j < w; j++){
			printf("%f  ", M[i*w+j]);
		}
		printf("\n");
	}
	printf("\n");
}

// whole training process
// X (d_X): pointer to training data, TRAINING_SIZE x TRAINING_DIM = 4 x 4
// X_w: training data width, or number of training data (TRAINING_DIM)
// X_h: training data height, or number of features (TRAINING_SIZE)
// y: pointer to labels of training data
// y_w: 1
// l1: pointer to Layer 1 output?, L1_SIZE x TRAINING_SIZE
// l1_w: layer 1 width, or L1_SIZE (8)
// l_1_d: pointer to layer 1 delta
// pred: pointer to prediction, e.g. pred[0] is prediction of 1st data
// pred_d: prediction delta
// W0: weight 0
// W1: weight 1
// buffer
/*
input X wiht 4 features -> W0 (8x4 matrix) -> hidden layer 1 with 8 neurons -> sigmoid -> W1 (1x8 matrix) -> output
*/
__global__ 
void kFit(	const float* X, const int X_w, const int X_h,
						const float* y, const int y_w,
						float* l1, const int l1_w, float* l_1_d,
						float* pred, float* pred_d,
						float* W0,
						float* W1,
						float* buffer
						)
{
	// print
	printf("W1\n");
	kPrintMatrix(W1, 1, 8);
	printf("W0\n");
	kPrintMatrix(W0, 8, 4);
	printf("\n");
	
	// for 50 iterations
	for (unsigned i = 0; i < 50; ++i) {
		// forward propagation
		// activate(input * weight) for input > layer 1 and for layer 1 > output
		/*
		line 1: 
			X = input, l1 = (X dot W0) = in_h1 (h1: hidden layer 1 )
			l1 = sigmoid(l1) = out_h1 (1 x 8)
		line 2: 
			pred = (1 dot W1) = IN_output, pred = sigmoid(pred) = output
		*/
        dSigmoid(dDot(X, W0, l1, X_h, X_w, l1_w), l1, X_h, l1_w);		
		dSigmoid(dDot(l1, W1, pred, X_h, l1_w, y_w), pred, X_h, y_w);
		
		// backpropagate errors
		/* 
		line 1: 
			x or *: elementwise matrix multiplication; dot: matrix dot product
			pred_d = y - pred (my: y_cap - y)
			buffer = sigmoid_d(pred)
			pred_d = pred_d x buffer = (y - y_cap) * y * (1 - y)
		line 2: 
			l_1_d = (pred_d dot W1_transpose)
			buffer = sigmoid_d(l1)
			l_1_d = l_1_d x buffer
		*/
        dMatrixByMatrixElementwise(dMatrixSubstractMatrix(y, pred, pred_d, X_h, y_w), dSigmoid_d(pred, buffer, X_h, y_w), pred_d, X_h, y_w );
		dMatrixByMatrixElementwise(dDot_m1_m2T(pred_d, W1, l_1_d, X_h, y_w, l1_w), dSigmoid_d(l1, buffer, X_h, l1_w), l_1_d, X_h, l1_w);

		// update weights 0 and 1
		/*
		line 1: W1 = (l1_transpose dot pred_d)
		line 2: W0 = (X_transpose dot l_1_d)
		*/
        dDot_m1T_m2( l1, pred_d, W1, X_h, l1_w, y_w );
		dDot_m1T_m2( X, l_1_d, W0, X_h, X_w, l1_w );
		// print
		printf("W1\n");
		kPrintMatrix(W1, 1, 8);
		printf("W0\n");
		kPrintMatrix(W0, 8, 4);
		printf("\n");
    }
}

int main(void){
	// h_: host (CPU)
	// d_: device (GPU)

	// Define hyperparameters
	const int TRAINING_SIZE = 4;	// number of samples
	const int TRAINING_DIM = 4;		// number of features
	const int L1_SIZE = 8;

	// X, the first 4 lines from Iris dataset
	// sepal length, sepal width, petal length, petal width
	// 4 points with 4 features each 
	float h_X[TRAINING_SIZE*TRAINING_DIM] = {	5.1, 3.5, 1.4, 0.2,
												4.9, 3.0, 1.4, 0.2,
												6.2, 3.4, 5.4, 2.3,
												5.9, 3.0, 5.1, 1.8 };
	
	// define input size
	const signed int X_size = sizeof(h_X);
	
	// allocate space on GPU for input data
	float *d_X;
	hipMalloc(&d_X, X_size);
	hipMemcpy(d_X, h_X, X_size, hipMemcpyHostToDevice);

	// initialize weight 0 randomly
	const long signed int W0_size = L1_SIZE*TRAINING_DIM*sizeof(float);
	float *h_W0 = (float*)malloc(W0_size);
	for (int i = 0; i < L1_SIZE*TRAINING_DIM; i++){
	    h_W0[i] = 0.1 * (2.0*rand()/RAND_MAX-1.0);
	}

	// allocate space on GPU for weight 0
	float *d_W0;
	hipMalloc(&d_W0, W0_size);
	hipMemcpy(d_W0, h_W0, W0_size, hipMemcpyHostToDevice);

	// layer 1 size
	// LAYER_1, LAYER_1_DELTA AND BUFFER OF LAYER 1 SIZE
	const long signed int L1_size = L1_SIZE*TRAINING_SIZE*sizeof(float);

	// define layer 1 variables (delta and buffer)
	float* h_layer_1 = (float*)malloc(L1_size);
	float* h_layer_1_delta = (float*)malloc(L1_size);
	float* h_buffer = (float*)malloc(L1_size);

	// give layer 1 initial 0 values
	for (int i = 0; i < L1_SIZE*TRAINING_SIZE; i++){
	    h_layer_1[i] = 0.0;
	    h_buffer[i] = 0.0;
	    h_layer_1_delta[i] = 0.0;
	}

	// allocate GPU space for layer 1
	float *d_layer_1;
	hipMalloc(&d_layer_1, L1_size);
	hipMemcpy(d_layer_1, h_layer_1, L1_size, hipMemcpyHostToDevice);

	// allocate GPU space for layer 1 buffer 
	float *d_buffer;
	hipMalloc(&d_buffer, L1_size);
	hipMemcpy(d_buffer, h_buffer, L1_size, hipMemcpyHostToDevice);

	// allocate GPU space for layer 1 data
	float *d_layer_1_delta;
	hipMalloc(&d_layer_1_delta, L1_size);
	hipMemcpy(d_layer_1_delta, h_layer_1_delta, L1_size, hipMemcpyHostToDevice);

	// allocate weight 1 randomly
	const long signed int W1_size = L1_SIZE*sizeof(float);
	float *h_W1 = (float*)malloc(W1_size);
	for (int i = 0; i < L1_SIZE; i++){
	    h_W1[i] = 0.1* (2.0*rand()/RAND_MAX-1.0);
	}

	// allocate GPU space for weight 1
	float *d_W1;
	hipMalloc(&d_W1, W1_size);
	hipMemcpy(d_W1, h_W1, W1_size, hipMemcpyHostToDevice);

	// 4 labels
	float h_y[4] = {	0,
						0,
						1,
						1 };

	// allocate GPU space for labels 
	const signed int y_size = sizeof(h_y);
	float *d_y;
	hipMalloc(&d_y, y_size);
	hipMemcpy(d_y, h_y, y_size, hipMemcpyHostToDevice);

	// define prediction and prediction delta
	float* h_pred = (float*)malloc(y_size);
	float* h_pred_delta = (float*)malloc(y_size);
	for (int i = 0; i < TRAINING_SIZE; i++){
	    h_pred[i] = 0.0;
	    h_pred_delta[i] = 0.0;
	}

	// allocate GPU space for predciction
	float *d_pred;
	hipMalloc(&d_pred, y_size);
	hipMemcpy(d_pred, h_pred, y_size, hipMemcpyHostToDevice);

	// allocate GPU space for prediction delta
	float *d_pred_delta;
	hipMalloc(&d_pred_delta, y_size);
	hipMemcpy(d_pred_delta, h_pred_delta, y_size, hipMemcpyHostToDevice);

	// train model on 1 block with 1 thread given initial vars
	kFit <<< 1, 1 >>> (	d_X, TRAINING_DIM, TRAINING_SIZE,
						d_y, 1,
						d_layer_1, L1_SIZE, d_layer_1_delta,
						d_pred,
						d_pred_delta,
						d_W0,
						d_W1,
						d_buffer);
	
	// send computed vars back to host
	hipMemcpy(h_pred, d_pred, y_size, hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(d_pred);
	hipFree(d_X);
	hipFree(d_y);
	hipFree(d_layer_1_delta);
	hipFree(d_pred_delta);
	hipFree(d_W0);
	hipFree(d_W1);
	hipFree(d_buffer);

	// free CPU memory
	free(h_layer_1_delta);
	free(h_pred_delta);
	free(h_W0);
	free(h_W1);
	free(h_buffer);

	// print prediction, label and error values
	for (int i = 0; i < TRAINING_SIZE; i++){
		printf("Prediction[%i] : %f True Value[%i] : %f Error[%i] : %f\n", i, h_pred[i], i, h_y[i], i, h_pred[i] - h_y[i]);
	}

	// free CPU memory of final prediction var
	free(h_pred);
}