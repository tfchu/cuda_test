#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__
void hello() 
{
    printf("Hello World! I'm a thread in block %d\n", blockIdx.x);
}

int main(int argc, char ***argv)
{
    // Launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    hipDeviceSynchronize();

    printf("That's all\n");

    return 0;
}