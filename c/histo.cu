#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int log2(int i)
{
    int r = 0;
    while (i >>= 1) r++;
    return r;
}

int bit_reverse(int w, int bits)
{
    int r = 0;
    for (int i = 0; i < bits; i++)
    {
        int bit = (w & (1 << i)) >> i;
        r |= bit << (bits - i - 1);
    }
    return r;
}

/*
Using device 0:
NVIDIA Tegra X1; global mem: 2076037120B; compute v5.3; clock: 921600 kHz
Running naive histo
bin 0: count 7
bin 1: count 7
bin 2: count 6
bin 3: count 6
bin 4: count 7
bin 5: count 6
bin 6: count 7
bin 7: count 6
bin 8: count 7
bin 9: count 7
bin 10: count 7
bin 11: count 7
bin 12: count 7
bin 13: count 6
bin 14: count 6
bin 15: count 8

incorrect due to race condition in d_bins[myBin]++
this does not happen in serial code as each thread runs separately
e.g. BIN with value 5, and thread 1 and 2 wants to increase it
    - thread 1 reads 5, increase to 6, write 6 back to bin
    - thread 2 reads 5, increase to 6, write 6 back to bin
    - but actual answer is 7
*/
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    d_bins[myBin]++;
}

/*
Using device 0:
NVIDIA Tegra X1; global mem: 2076037120B; compute v5.3; clock: 921600 kHz
Running simple histo
bin 0: count 4096
bin 1: count 4096
bin 2: count 4096
bin 3: count 4096
bin 4: count 4096
bin 5: count 4096
bin 6: count 4096
bin 7: count 4096
bin 8: count 4096
bin 9: count 4096
bin 10: count 4096
bin 11: count 4096
bin 12: count 4096
bin 13: count 4096
bin 14: count 4096
bin 15: count 4096
*/
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    atomicAdd(&(d_bins[myBin]), 1);
}


int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate);
    }

    const int ARRAY_SIZE = 65536;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
    const int BIN_COUNT = 16;
    const int BIN_BYTES = BIN_COUNT * sizeof(int);

    // generate the input array on the host
    int h_in[ARRAY_SIZE];
    for(int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = bit_reverse(i, log2(ARRAY_SIZE));
    }
    int h_bins[BIN_COUNT];
    for(int i = 0; i < BIN_COUNT; i++) {
        h_bins[i] = 0;
    }

    // declare GPU memory pointers
    int * d_in;
    int * d_bins;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_bins, BIN_BYTES);

    // transfer the arrays to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 
    hipMemcpy(d_bins, h_bins, BIN_BYTES, hipMemcpyHostToDevice); 

    int whichKernel = 0;
    if (argc == 2) {
        whichKernel = atoi(argv[1]);
    }
        
    // launch the kernel
    switch(whichKernel) {
    case 0:
        printf("Running naive histo\n");
        naive_histo<<<ARRAY_SIZE / 64, 64>>>(d_bins, d_in, BIN_COUNT);
        break;
    case 1:
        printf("Running simple histo\n");
        simple_histo<<<ARRAY_SIZE / 64, 64>>>(d_bins, d_in, BIN_COUNT);
        break;
    default:
        fprintf(stderr, "error: ran no kernel\n");
        exit(EXIT_FAILURE);
    }

    // copy back the sum from GPU
    hipMemcpy(h_bins, d_bins, BIN_BYTES, hipMemcpyDeviceToHost);

    for(int i = 0; i < BIN_COUNT; i++) {
        printf("bin %d: count %d\n", i, h_bins[i]);
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_bins);
        
    return 0;
}